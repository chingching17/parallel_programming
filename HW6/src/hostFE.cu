#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "helper.h"
extern "C"{
#include "hostFE.h"
}

__global__ void convolution(int filterWidth, float *filter, int imageHeight, int imageWidth, float *inputImage, float *outputImage){
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;

    int idx = thisX + thisY * imageWidth;

    int half_filterSize = filterWidth / 2;
    float sum = 0;
    int k, l; 
    for(k = -half_filterSize; k <= half_filterSize; k++){
        for(l = -half_filterSize; l <= half_filterSize; l++){
            if(thisY + k >= 0 && thisY + k < imageHeight && thisX + l >= 0 && thisX + l < imageWidth){
                sum += inputImage[(thisY + k) * imageWidth + thisX + l] * filter[(k + half_filterSize) * filterWidth + l + half_filterSize];
            }

        }
    }
    outputImage[idx] = sum;
}

void hostFE(int filterWidth, float *filter, int imageHeight, int imageWidth,
            float *inputImage, float *outputImage, cl_device_id *device,
            cl_context *context, cl_program *program)
{
    // set size
    int filterSize = filterWidth * filterWidth;
    int mem_size = imageHeight * imageWidth;

    // allocate memory
    float *data_filter, *data_inputImage, *data_outputImage;
    hipMalloc(&data_filter, filterSize * sizeof(float));
    hipMalloc(&data_inputImage, mem_size * sizeof(float));
    hipMalloc(&data_outputImage, mem_size * sizeof(float));

    // copy host to cuda device
    hipMemcpy(data_filter, filter, filterSize * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(data_inputImage, inputImage, mem_size * sizeof(float), hipMemcpyHostToDevice);

    // run CUDA
    dim3 threadsPerBlock(25, 25);
    dim3 numBlocks(imageWidth / threadsPerBlock.x, imageHeight / threadsPerBlock. y);
    convolution<<<numBlocks, threadsPerBlock>>>(filterWidth, data_filter, imageHeight, imageWidth, data_inputImage, data_outputImage);

    // copy result to host
    hipMemcpy(outputImage, data_outputImage, mem_size * sizeof(float), hipMemcpyDeviceToHost);

    // free memory
    hipFree(data_filter);
    hipFree(data_inputImage);
    hipFree(data_outputImage);
}