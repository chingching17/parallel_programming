#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int width, int *data_img, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    float x = lowerX + thisX * stepX;
    float y = lowerY + thisY * stepY;

    float tmpX = x;
    float tmpY = y;
    int idx = thisX + thisY * width;
    for(int i = 0; i < maxIterations; i++){
        if (tmpX * tmpX + tmpY * tmpY > 4.f)break;
        float new_x = tmpX * tmpX - tmpY * tmpY;
        float new_y = 2.f * tmpX * tmpY;
        tmpX = x + new_x;
        tmpY = y + new_y;
    }
    data_img[idx] = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int *result_img, *data_img;
    result_img = (int *)malloc(resX * resY * sizeof(int));
    hipMalloc((void **)&data_img, resX * resY * sizeof(int));

    dim3 ThreadsPerBlock(16, 16);
    dim3 NumOfBlocks(resX / ThreadsPerBlock.x, resY / ThreadsPerBlock.y);
    mandelKernel<<<NumOfBlocks, ThreadsPerBlock>>>(lowerX, lowerY, stepX, stepY, resX, data_img, maxIterations);

}
