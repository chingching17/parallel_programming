#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void mandelKernel(float lowerX, float lowerY, float stepX, float stepY, int width, int *data_img, int maxIterations, size_t pitch) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;
    int thisX = blockIdx.x * blockDim.x + threadIdx.x;
    int thisY = blockIdx.y * blockDim.y + threadIdx.y;
    float x = lowerX + thisX * stepX;
    float y = lowerY + thisY * stepY;

    float tmpX = x;
    float tmpY = y;
    for(int i = 0; i < maxIterations; i++){
        if (tmpX * tmpX + tmpY * tmpY > 4.f)break;
        float new_x = tmpX * tmpX - tmpY * tmpY;
        float new_y = 2.f * tmpX * tmpY;
        tmpX = x + new_x;
        tmpY = y + new_y;
    }
    int* row = (int *)((char*)data_img + thisY * pitch);
    row[thisX] = i;
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
    float stepX = (upperX - lowerX) / resX;
    float stepY = (upperY - lowerY) / resY;

    int *result_img, *data_img;
    hipHostAlloc((void **)&result_img, resX * resY * sizeof(int), hipHostMallocDefault);

    size_t pitch;
    hipMallocPitch((void **)&data_img, &pitch, resX * sizeof(int), resY);

    dim3 ThreadsPerBlock(16, 16);
    dim3 NumOfBlocks(resX / ThreadsPerBlock.x, resY / ThreadsPerBlock.y);
    mandelKernel<<<NumOfBlocks, ThreadsPerBlock>>>(lowerX, lowerY, stepX, stepY, resX, data_img, maxIterations, pitch);

    hipMemcpy2D(result_img, resX * sizeof(int), data_img, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);
    memcpy(img, result_img, resX * resY * sizeof(int));

    hipHostFree(result_img);
    hipFree(data_img);
}
